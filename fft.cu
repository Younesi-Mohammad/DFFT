#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

__device__ unsigned int RightIndex(unsigned int num,unsigned int M ) //making right indexes
{ 
	unsigned int right_num = 0; 
	int temp;
	
    for ( int i = 0; i < M; i++)
    {
        temp = (num & (1 << i));
        if(temp){
			right_num = right_num |(1 << ((M - 1) - i));
		}
    }
  
    return right_num;
}

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M,unsigned int i) 
{   

    float tempr,tempi;
	
	unsigned int ind = bx*blockDim.x+tx;
    unsigned int k_butt = ind%(1<<(M-1-i));
    unsigned int k_part = (unsigned int)(ind/(1<<(M-1-i)));
    unsigned int butter_ind = 2*(1<<i)*k_butt + k_part;
    unsigned int w = k_part*(1<<(M-1-i));
  
           
    if (i==0) {
		
		tempr = cos((2*PI*w)/N)*x_r_d[RightIndex(butter_ind + 1<<i,M)] + sin((2*PI*w)/N)*x_i_d[RightIndex(butter_ind + 1<<i,M)];
		tempi = cos((2*PI*w)/N)*x_i_d[RightIndex(butter_ind + 1<<i,M)] - sin((2*PI*w)/N)*x_r_d[RightIndex(butter_ind + 1<<i,M)];
	 
		X_r_d[butter_ind] = x_r_d[RightIndex(butter_ind,M)] + tempr;
		X_i_d[butter_ind] = x_i_d[RightIndex(butter_ind,M)] + tempi;
		
		X_r_d[butter_ind + 1<<i] = x_r_d[RightIndex(butter_ind,M)] - tempr;
		X_i_d[butter_ind + 1<<i] = x_i_d[RightIndex(butter_ind,M)] - tempi;
    }
    else {
		
		tempr = cos((2*PI*w)/N)*x_r_d[butter_ind + 1<<i] + sin((2*PI*w)/N)*x_i_d[butter_ind + 1<<i]; //real part of multiplication
		tempi = cos((2*PI*w)/N)*x_i_d[butter_ind + 1<<i] - sin((2*PI*w)/N)*x_r_d[butter_ind + 1<<i]; //imaginary part of multiplication
	 
		X_r_d[butter_ind] = x_r_d[butter_ind] + tempr;
		X_i_d[butter_ind] = x_i_d[butter_ind] + tempi;
		
		X_r_d[butter_ind + 1<<i] = x_r_d[butter_ind] - tempr;
		X_i_d[butter_ind + 1<<i] = x_i_d[butter_ind] - tempi; 
    }
    

}
//-----------------------------------------------------------------------------
__global__ void kernelFunc2(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M) 
{
	//...
}
//-----------------------------------------------------------------------------
void gpuKernel_simple(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.

    int a,b;
    
    float* image,*real;
    real = (float*) malloc(N * sizeof(float));
    image = (float*) malloc(N * sizeof(float));
    
    
    if (M>=11) { a=1024; b=N/2048;}
    else {a=N/2;b=1;}

	dim3 dimGrid(b,1,1);
	dim3 dimBlock(a,1,1);
	int k;

    for(k=0;k<M;++k){
        if (k % 2 == 0 ){
        kernelFunc <<< dimGrid, dimBlock >>> (x_r_d, x_i_d, X_r_d, X_i_d, N, M,k);
        
        HANDLE_ERROR(hipMemcpy(real, X_r_d, N * sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(image, X_i_d, N * sizeof(float), hipMemcpyDeviceToHost));
        
        }
        else{
        kernelFunc <<< dimGrid, dimBlock >>> (X_r_d, X_i_d, x_r_d, x_i_d, N, M,k);
        
        HANDLE_ERROR(hipMemcpy(real, x_r_d, N * sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(image, x_i_d, N * sizeof(float), hipMemcpyDeviceToHost));
        
        }
    }
    if (M%2==0){
    HANDLE_ERROR(hipMemcpy(X_r_d, x_r_d, N * sizeof(float), hipMemcpyDeviceToDevice));
    HANDLE_ERROR(hipMemcpy(X_i_d, x_i_d, N * sizeof(float), hipMemcpyDeviceToDevice));    
    }

    HANDLE_ERROR(hipMemcpy(real, X_r_d, N * sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(image, X_i_d, N * sizeof(float), hipMemcpyDeviceToHost));
	free(real);
	free(image);

}
//-----------------------------------------------------------------------------
void gpuKernel_efficient(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	
	dim3 dimGrid(1,1);
	dim3 dimBlock(1,1);

	kernelFunc2 <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M);
}
